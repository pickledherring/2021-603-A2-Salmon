#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <tuple>
#include <iostream>
#include <semaphore.h>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <bits/stdc++.h>

using namespace std;

__global__ void KNN(float* test, float* train, float* predictions, int k,  int n_test, int n_train, int n_classes) {
    // Implements a parallel kNN where for each candidate query an in-place priority queue is maintained to identify the kNN's.
    float distance(float* a, float* b) {
        float sum = 0;
        for (int i = 0; i < n_classes - 1; i++) {
            float diff = a[i] - b[i];
            sum += diff * diff;
        }
        return sum;
    }

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // stores k-NN candidates for a query vector as a sorted 2d array. First element is inner product, second is class.
    float* candidates = (float*)calloc(k * 2, sizeof(float));
    for (int i = 0; i < 2 * k; i++) {candidates[i] = FLT_MAX;}
    // Stores bincounts of each class over the final set of candidate NN
    int* classCounts = (int*)calloc(n_classes, sizeof(int));

    if (tid < n_test) {
        for (int keyIndex = 0; keyIndex < n_train; keyIndex++) {
            float dist = distance(test[tid], train[key_Index]);
            // Add to our candidates
            for(int c = 0; c < k; c++){
                if(dist < candidates[2 * c]){
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for(int x = k - 2; x >= c; x--) {
                        candidates[2 * x + 2] = candidates[2 * x];
                        candidates[2 * x + 3] = candidates[2 * x + 1];
                    }
                    
                    // Set key vector as potential k NN
                    candidates[2 * c] = dist;
                    // class value
                    candidates[2 * c + 1] = train[keyIndex][n_classes - 1];
                    break;
                }
            }
        }

        // Bincount the candidate labels and pick the most common
        for(int i = 0; i < k; i++) {
            classCounts[(int)candidates[2 * i + 1]] += 1;
        }
        
        int max = -1;
        int max_index = 0;
        for (int i = 0; i < n_classes; i++) {
            if (classCounts[i] > max){
                max = classCounts[i];
                max_index = i;
            }
        }

        predictions[tid] = max_index;
        for (int i = 0; i < 2 * k; i++) {candidates[i] = FLT_MAX;}
        memset(classCounts, 0, n_classes * sizeof(int));
    }
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset) {
    // matrix size numberClasses x numberClasses
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int));
    
    for (int i = 0; i < dataset->num_instances(); i++) {
        // for each instance compare the true class and predicted class
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset) {
    int successfulPredictions = 0;
    
    for(int i = 0; i < dataset->num_classes(); i++) {
        // elements in the diagonal are correct predictions
        successfulPredictions += confusionMatrix[i * dataset->num_classes() + i];
    }
    
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        cout << "Usage: ./main datasets/train.arff datasets/test.arff k" << endl;
        exit(0);
    }

    int k = strtol(argv[3], NULL, 10);

    // Open the datasets
    ArffParser parserTrain(argv[1]);
    ArffParser parserTest(argv[2]);
    ArffData* train = parserTrain.parse();
    ArffData* test = parserTest.parse();
    // predictions is the array where you have to return the class predicted (integer) for the test dataset instances
    int* predictions;
    float* test_floats, * train_floats;
    for (int i = 0; i < test->num_instances(); i++) {
        for (int j = 0; j < test->num_attributes(); j++) {
            test_floats[i][j] = test->get_instance(i)->get(j)->operator float();
        }
    }
    for (int i = 0; i < train->num_instances(); i++) {
        for (int j = 0; j < train->num_attributes(); j++) {
           train_floats[i][j] = train->get_instance(i)->get(j)->operator float();
        }
    }

    float* d_test_floats, * d_train_floats;
    int* d_predictions;

    hipMalloc(&d_train_floats, train->num_attributes * train->num_instances() * sizeof(float));
    hipMalloc(&d_test_floats, test->num_attributes * test->num_instances() * sizeof(float));
    hipMalloc(&d_predictions, test->num_instances() * sizeof(int));

    hipMemcpy(d_test_floats, test_floats, test->num_attributes * test->num_instances() * sizeof(float),
                            cudaMemcpyHosttoDevice);
    hipMemcpy(d_train_floats, train_floats, train->num_attributes * train->num_instances() * sizeof(float),
                            cudaMemcpyHosttoDevice);

    int threads_per_block = 64;
    int grid_size = (test->num_instances() + threads_per_block + 1) / threads_per_block;

    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    KNN<<<grid_size, threads_per_block>>>(d_test_floats, d_train_floats, d_predictions, k, test->num_instances());
    hipMemcpy(predictions, d_predictions, test->num_instances() * sizeof(int), hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    // Compute the confusion matrix
    int* confusionMatrix = computeConfusionMatrix(predictions, test);
    // Calculate the accuracy
    float accuracy = computeAccuracy(confusionMatrix, test);

    hipFree(d_test_floats);
    hipFree(d_train_floats);
    hipFree(d_predictions);

    free(test_floats);
    free(train_floats);
    free(predictions);

    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

    printf("The %i-NN classifier for %lu test instances on %lu train instances required %llu ms CPU time. Accuracy was %.4f\n",
                    k, test->num_instances(), train->num_instances(), (long long unsigned int) diff, accuracy);
}
