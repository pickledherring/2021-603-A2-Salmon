#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <limits.h>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"

__global__ void KNN(float* test, float* train, int* predictions, float* candidates, int* classCounts,
					int k,  int n_test, int n_train, int n_att, int n_classes) {
    // Implements a parallel kNN where for each candidate query an
	// in-place priority queue is maintained to

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < (2 * k); i++) {
        		candidates[i] = FLT_MAX;
    }

    if (tid < n_test) {
    	if (tid == 64) {
    		for (int i = 0; i < n_train * n_att; i++) {
    			printf("train[%d]: %f ", i, train[i]);
    		}
    	}
        for (int keyIndex = 0; keyIndex < n_train; keyIndex++) {
//        	if (tid == 64) {printf("\tkey index is %d\n", keyIndex);}
        	float dist = 0;
			for (int i = 0; i < n_att - 1; i++) {
//				if (tid == 64) {printf("\ti is %d\n", i);}

				float diff = test[tid * n_att + i] - train[keyIndex * n_att + i];
				dist += diff * diff;
//				if (tid == 64) {printf("\t\tdist = %f \n", dist);}
			}
            // Add to our candidates
            for (int c = 0; c < k; c++){
//            	if (tid == 64) {printf("\t\t%f < %f?: \n", dist, candidates[2 * c]);}
                if (dist < candidates[2 * c]){
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for (int x = k - 2; x >= c; x--) {
                        candidates[2 * x + 2] = candidates[2 * x];
                        candidates[2 * x + 3] = candidates[2 * x + 1];
                        if (tid == 64) {
                        	printf("candidates[2 * %d + 2] = %f\n", x, candidates[2 * x]);
                        	printf("candidates[2 * %d + 3] = %f\n", x, candidates[2 * x + 1]);
                        }
                    }
                    
                    // Set key vector as potential k NN
                    candidates[2 * c] = dist;
                    // class value
                    candidates[2 * c + 1] = train[keyIndex * n_att - 1];
                    if (tid == 64) {printf("candidates[%d] = %f\n", 2 * c + 1,
                    		train[keyIndex * n_att - 1]);}
                    break;
                }
            }
            printf("made it to d! rank %d", tid);
        // Bincount the candidate labels and pick the most common
        for (int i = 0; i < k; i++) {
        	if (tid == 64) {printf("(int)candidates[2 * %d + 1]: %d\n", i, candidates[2 * i + 1]);}
            classCounts[(int)candidates[2 * i + 1]] += 1;
        }
        printf("made it to e! rank %d", tid);

        
        int max = -1;
        int max_index = 0;
        for (int i = 0; i < n_classes; i++) {
            if (classCounts[i] > max){
                max = classCounts[i];
                max_index = i;
            }
        }
//        if (tid == 0 && keyIndex == n_train - 1) {printf("made it to e!\n");}
        predictions[tid] = max_index;
        for (int i = 0; i < 2 * k; i++) {candidates[i] = FLT_MAX;}
        memset(classCounts, 0, n_classes * sizeof(int));
        }
    }
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset) {
    // matrix size numberClasses x numberClasses
    int* confusionMatrix = (int*)malloc(dataset->num_classes() *
    		dataset->num_classes() * sizeof(int));
    printf("made it to f!\n");
    for (int i = 0; i < dataset->num_instances(); i++) {
        // for each instance compare the true class and predicted class
        int trueClass = dataset->get_instance(i)->get(
        		dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    printf("made it to g!\n");
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset) {
    int successfulPredictions = 0;
    printf("made it to h!\n");
    for (int i = 0; i < dataset->num_classes(); i++) {
        // elements in the diagonal are correct predictions
        successfulPredictions += confusionMatrix[i * dataset->num_classes() + i];
    }
    printf("made it to i!\n");
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: ./KNN datasets/train.arff datasets/test.arff k\n");
        exit(0);
    }
    int k = strtol(argv[3], NULL, 10);
    // Open the datasets
    ArffParser parserTrain(argv[1]);
    ArffParser parserTest(argv[2]);
    ArffData* train = parserTrain.parse();
    ArffData* test = parserTest.parse();
    int test_num = test->num_instances();
    int train_num = train->num_instances();
    int att_num = train->num_attributes();
    int n_classes = train->num_classes();
    printf("made it to k!\n");
    // predictions is the array where you have to return the class
    	// predicted (integer) for the test dataset instances
    int* h_predictions = (int*)malloc(test_num * sizeof(int));
    float* h_test_floats = (float*)malloc(test_num * att_num * sizeof(float));
    float* h_train_floats = (float*)malloc(train_num * att_num * sizeof(float));
    for (int i = 0; i < test_num; i++) {
        for (int j = 0; j < att_num; j++) {
            h_test_floats[i * att_num + j] = test->get_instance(i)->get(j)
            										->operator float();
        }
    }
    printf("made it to l!\n");
    for (int i = 0; i < train_num; i++) {
        for (int j = 0; j < att_num; j++) {
           h_train_floats[i * att_num + j] = train->get_instance(i)->get(j)
        											->operator float();
        }
    }
    printf("made it to m!\n");
    float* d_test_floats, * d_train_floats, * candidates;
    int* classCounts, * d_predictions;

    hipMalloc(&d_train_floats, train_num * att_num * sizeof(float));
    hipMalloc(&d_test_floats, test_num * att_num * sizeof(float));
    hipMalloc(&d_predictions, test_num * sizeof(int));
    hipMalloc(&candidates, k * 2 * sizeof(float));
    hipMalloc(&classCounts, test_num * sizeof(int));
    printf("made it to n!\n");
    hipMemcpy(d_test_floats, h_test_floats, att_num * test_num * sizeof(float),
    		hipMemcpyHostToDevice);
    hipMemcpy(d_train_floats, h_train_floats, att_num * train_num * sizeof(float),
    		hipMemcpyHostToDevice);
    int threads_per_block = 128;
    int grid_size = (test_num + threads_per_block - 1) / threads_per_block;
    printf("made it to o!\n");
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    KNN<<<grid_size, threads_per_block>>>(d_test_floats, d_train_floats, d_predictions,
    		candidates, classCounts, k, test_num, train_num, att_num, n_classes);
    hipMemcpy(h_predictions, d_predictions, test_num * sizeof(int),
    		hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    printf("made it to o!\n");

    // Compute the confusion matrix
    int* confusionMatrix = computeConfusionMatrix(h_predictions, test);
    // Calculate the accuracy
    float accuracy = computeAccuracy(confusionMatrix, test);
    hipFree(d_test_floats);
    hipFree(d_train_floats);
    hipFree(d_predictions);
    hipFree(d_train_floats);

    free(h_test_floats);
    free(h_train_floats);
    free(h_predictions);
    hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t,
				hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) +
    		end.tv_nsec - start.tv_nsec) / 1e6;

    printf("The %i-NN classifier for %d test instances on"
    		" %d train instances required %llu ms CPU time. Accuracy was %.4f\n",
            k, test_num, train_num, (long long unsigned int)diff, accuracy);
}
